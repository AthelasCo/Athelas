#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__device__ inline int updiv(int n, int d) {
    return (n+d-1)/d;
}


static inline int updivHost(int n, int d) {
    return (n+d-1)/d;
}

struct GlobalConstants {

    double* cudaConstantProbTable;
    unsigned long long cudaDeviceNumEdges, cudaDeviceNumVertices;
};

__constant__ GlobalConstants cuConstGraphParams;


bool setup(
        const unsigned long long nEdges,
        const unsigned long long nVertices,
        const double RMAT_a, const double RMAT_b, const double RMAT_c,
        const unsigned int nCPUWorkerThreads,
        std::ofstream& outFile,
        const unsigned long long standardCapacity,
        const bool allowEdgeToSelf,
        const bool allowDuplicateEdges,
        const bool directedGraph,
        const bool sorted){
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 480") == 0
            || name.compare("GeForce GTX 670") == 0
            || name.compare("GeForce GTX 780") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 480, 670 or 780.\n");
        printf("---------------------------------------------------------\n");
    }
    double* cudaConstantProbTable;
    static std::default_random_engine generator;

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy
    hipMalloc(&cudaConstantProbTable , sizeof(double) * 128 * 4 );
    // hipMemcpy(&cudaDeviceNumEdges, sizeof(unsigned long long));
    // hipMemcpy(&cudaDeviceNumVertices , sizeof(unsigned long long));


    GlobalConstants params;
    params.cudaConstantProbTable = cudaConstantProbTable;
    params.cudaDeviceNumEdges = nEdges ;
    params.cudaDeviceNumVertices = nVertices;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstGraphParams), &params, sizeof(GlobalConstants));
    // hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    // hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    // hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    // hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    // hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);
    // hipMalloc(&cudaDeviceCircleList, sizeof(float) * (numCircles+THREADS_PER_BLOCK) * updivHost(image->width,NUM_THREADS) * updivHost(image->height,NUM_THREADS));

    // hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    // hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    // hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    // hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // hipMemset(cudaDeviceCircleList,0.f, sizeof(float) * numCircles * updivHost(image->width,NUM_THREADS) * updivHost(image->height,NUM_THREADS));

    // // Initialize parameters in constant memory.  We didn't talk about
    // // constant memory in class, but the use of read-only constant
    // // memory here is an optimization over just sticking these values
    // // in device global memory.  NVIDIA GPUs have a few special tricks
    // // for optimizing access to constant memory.  Using global memory
    // // here would have worked just as well.  See the Programmer's
    // // Guide for more information about constant memory.

    // GlobalConstants params;
    // params.sceneName = sceneName;
    // params.numCircles = numCircles;
    // params.imageWidth = image->width;
    // params.imageHeight = image->height;
    // params.position = cudaDevicePosition;
    // params.velocity = cudaDeviceVelocity;
    // params.color = cudaDeviceColor;
    // params.radius = cudaDeviceRadius;
    // params.imageData = cudaDeviceImageData;
    // params.circleList = cudaDeviceCircleList;

    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // // also need to copy over the noise lookup tables, so we can
    // // implement noise on the GPU
    // int* permX;
    // int* permY;
    // float* value1D;
    // getNoiseTables(&permX, &permY, &value1D);
    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // // last, copy over the color table that's used by the shading
    // // function for circles in the snowflake demo

    // float lookupTable[COLOR_MAP_SIZE][3] = {
    //     {1.f, 1.f, 1.f},
    //     {1.f, 1.f, 1.f},
    //     {.8f, .9f, 1.f},
    //     {.8f, .9f, 1.f},
    //     {.8f, 0.8f, 1.f},
    // };

    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
    return true;
}
