#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <iostream>
#include <cstring>
#include <fstream>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "GraphGen_notSorted_Cuda.h"

struct GlobalConstants {

    unsigned long long cudaDeviceNumEdges, cudaDeviceNumVertices;
    double* cudaDeviceProbs;
    unsigned* cudaDeviceOutput;
};

__device__ inline int updiv(int n, int d) {
    return (n+d-1)/d;
}

__constant__ GlobalConstants cuConstGraphParams;

__device__ __inline__ int2
get_Edge_indices( double offX, unsigned long long rngX, unsigned long long offY, unsigned long long rngY,  std::ref(distribution), std::ref(generator),double A[],double B[],double C[],double D[]) {
    int depth =0;
    double sumA, sumAB, sumABC, sumAC;
    while (rngX > 1 || rngY > 1) {
        sumA = A[depth];
        sumAB = sumA + B[depth];
        sumAC = sumA + C[depth];
        sumABC = sumAB + C[depth];
        const double RndProb = distribution(generator);

        if (rngX>1 && rngY>1) {
          if (RndProb < sumA) { rngX/=2; rngY/=2; }
          else if (RndProb < sumAB) { offX+=rngX/2;  rngX-=rngX/2;  rngY/=2; }
          else if (RndProb < sumABC) { offY+=rngY/2;  rngX/=2;  rngY-=rngY/2; }
          else { offX+=rngX/2;  offY+=rngY/2;  rngX-=rngX/2;  rngY-=rngY/2; }
        } else
        if (rngX>1) { // row vector
          if (RndProb < sumAC) { rngX/=2; rngY/=2; }
          else { offX+=rngX/2;  rngX-=rngX/2;  rngY/=2; }
        } else
        if (rngY>1) { // column vector
          if (RndProb < sumAB) { rngX/=2; rngY/=2; }
          else { offY+=rngY/2;  rngX/=2;  rngY-=rngY/2; }
        } else{
            std::cout<<"OH NO!\n";

        }
        depth++;
      }
    int2 e;
    e.x = offX;
    e.y = offY;
    return e;
}
__global__ void KernelGenerateEdges(const bool directedGraph,
        const bool allowEdgeToSelf, const bool sorted) {
    // std::uniform_int_distribution<>& dis, std::mt19937_64& gen,
    // std::vector<unsigned long long>& duplicate_indices
    int blockIndex = blockIdx.x;
    int threadIndex = threadIdx.x;
    specialSqaure squ = cuConstGraphParams.squares[blockIndex];
    __shared__ unsigned long long offX;  
    __shared__ unsigned long long offY;  
    __shared__ unsigned long long rngX;  
    __shared__ unsigned long long rngY;  
    
    unsigned long long nEdgesToGen = squ.getnEdges();

    std::default_random_engine generator;

    std::uniform_real_distribution<double> distribution(0.0,1.0);
    __shared__ double A[MAX_DEPTH];
    __shared__ double B[MAX_DEPTH];
    __shared__ double C[MAX_DEPTH];
    __shared__ double D[MAX_DEPTH];

    if (threadIndex==0)
    {
        for (int i = 0; i < MAX_DEPTH; ++i)
        {
            double4 prob= *(double4*)(&cuConstGraphParams.cudaDeviceProbs[4 * (i)]);
            A[i] = prob.x;
            B[i] = prob.y;
            C[i] = prob.z;
            D[i] = prob.w;
            offX = squ.p_x_start;
            offY = squ.squ.p_y_start;
            rngX = squ.get_X_end()-offX;
            rngY = squ.get_Y_end()-offY;
        }

    }

    auto applyCondition = directedGraph || ( squ.offX < squ.offY); // true: if the graph is directed or in case it is undirected, the square belongs to the lower triangle of adjacency matrix. false: the diagonal passes the rectangle and the graph is undirected.


    unsigned maxIter = updiv(nEdgesToGen, THREADS_PER_BLOCK);

    for (unsigned i = 0; i < maxIter; ++i)
    {
        int edgeIdx = i * THREADS_PER_BLOCK + threadIndex;
        int2 e;
        if (edgeIdx < nEdgesToGen )
        {

            while(true) {
                e = get_Edge_indices(offX, rngX, offY, rngY,  std::ref(distribution), std::ref(generator), A, B, C, D );
                unsigned long long h_idx = e.x+squ.p_x_start;
                unsigned long long v_idx = e.y+squ.p_y_start;
                if( (!applyCondition && h_idx > v_idx) || (!allowEdgeToSelf && h_idx == v_idx ) ) // Short-circuit if it doesn't pass the test.
                    continue;
                if (h_idx< offX || h_idx>= offX+ offY || v_idx < offY || v_idx >= offY+rngY )
                    continue;
                break;
            }
            cuConstGraphParams.cudaDeviceOutput[2*( squ.get_output_idx() + i*THREADS_PER_BLOCK + threadIndex )] = e;
        }
        __syncthreads();
    }

    // short imageWidth = cuConstRendererParams.imageWidth;
    // short imageHeight = cuConstRendererParams.imageHeight;
    // int numCircles = cuConstRendererParams.numCircles;

    // float invWidth = 1.f / imageWidth;
    // float invHeight = 1.f / imageHeight;

    //  __shared__ uint shared_no_of_circles[THREADS_PER_BLOCK];
    //  __shared__ uint shared_output[THREADS_PER_BLOCK];
    // volatile __shared__ uint shared_scratch[2 * THREADS_PER_BLOCK];
    // volatile __shared__ uint shared_circle_index[THREADS_PER_BLOCK];
    // __shared__ float3 position[THREADS_PER_BLOCK];
    // __shared__ float radii[THREADS_PER_BLOCK];
    // __shared__ float3 colors[THREADS_PER_BLOCK];
    // int circlesPerThread = updiv(numCircles,THREADS_PER_BLOCK);
    // float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
    //                                                  invHeight * (static_cast<float>(pixelY) + 0.5f));
    // float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + pixelX)]);
    // float4 existingColor = *imgPtr;
    // for (int i=0; i < circlesPerThread; i++) {
    //     int cIdx = i * THREADS_PER_BLOCK + threadIndex;
    //     shared_no_of_circles[threadIndex] = 0;

    //     if (cIdx < numCircles) {
    //         int cIdx3 = 3 * cIdx;
    //         float3 p = *(float3*)(&cuConstRendererParams.position[cIdx3]);
    //         float  rad = cuConstRendererParams.radius[cIdx];
    //         short minX = static_cast<short>(imageWidth * (p.x - rad));
    //         short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    //         short minY = static_cast<short>(imageHeight * (p.y - rad));
    //         short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;


    //             if(!(blockXmin > maxX || blockXmax < minX
    //             || blockYmin > maxY || blockYmax < minY)){
    //                 shared_no_of_circles[threadIndex]=1;
    //                 radii[threadIndex] = rad;
    //                 position[threadIndex] = p;
    //                 colors[threadIndex] = *(float3*)(&cuConstRendererParams.color[cIdx3]);
    //             }
            
    //     } 

    //     __syncthreads();

    //     sharedMemExclusiveScan(threadIndex, shared_no_of_circles, shared_output,
    //                           shared_scratch, THREADS_PER_BLOCK);

    //     __syncthreads();

    //     int numOverBlkCircles = shared_output[THREADS_PER_BLOCK - 1];
    //     if ( shared_no_of_circles[THREADS_PER_BLOCK - 1] == 1 )
    //         numOverBlkCircles += 1;

    //     if ( shared_no_of_circles[threadIndex] == 1 ) {
    //         shared_circle_index[shared_output[threadIndex]] = threadIndex;
    //     }

    //     __syncthreads();
        
    //     for (int j=0; j < numOverBlkCircles; j++) {
    //         int index = i * THREADS_PER_BLOCK + shared_circle_index[j];
    //             float3 p = position[shared_circle_index[j]];
    //             float rad = radii[shared_circle_index[j]];
    //             float3 color = colors[shared_circle_index[j]];
    //             shadePixel(pixelCenterNorm, p, &existingColor,rad, color);
    //     }


    // }
    // *imgPtr = existingColor;


}

////////////////////////////////////////////////////////////////////////////////////////



static inline int updivHost(int n, int d) {
    return (n+d-1)/d;
}


bool setup(
        const unsigned long long nEdges,
        const unsigned long long nVertices,
        const double RMAT_a, const double RMAT_b, const double RMAT_c,
        const unsigned int nCPUWorkerThreads,
        std::ofstream& outFile,
        const unsigned long long standardCapacity,
        const bool allowEdgeToSelf,
        const bool allowDuplicateEdges,
        const bool directedGraph,
        const bool sorted){
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 480") == 0
            || name.compare("GeForce GTX 670") == 0
            || name.compare("GeForce GTX 780") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 480, 670 or 780.\n");
        printf("---------------------------------------------------------\n");
    }
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy
    double* cudaDeviceProbs = NULL;
    hipMalloc(&cudaDeviceProbs, sizeof(double) * 4 * MAX_DEPTH);

    unsigned* cudaDeviceOutput = NULL;
    hipMalloc(&cudaDeviceOutput, sizeof(unsigned) * 2 * nEdges);

    GlobalConstants params;
    // // Initialize parameters in constant memory.  We didn't talk about
    // // constant memory in class, but the use of read-only constant
    // // memory here is an optimization over just sticking these values
    // // in device global memory.  NVIDIA GPUs have a few special tricks
    // // for optimizing access to constant memory.  Using global memory
    // // here would have worked just as well.  See the Programmer's
    // // Guide for more information about constant memory.


    //Generate Probabilities
    std::uniform_real_distribution<double> distribution(0.0,1.0);
    static std::default_random_engine generator;
    double probs[MAX_DEPTH*4];
    for (int i = 0; i < MAX_DEPTH*4; i+=4) {
        double A = RMAT_a * (distribution(generator)+0.5);
        double B = RMAT_b * (distribution(generator)+0.5);
        double C = RMAT_c *(distribution(generator)+0.5);
        double D = (1- (RMAT_a+RMAT_b+RMAT_c)) *(distribution(generator)+0.5);
        double abcd = A+B+C+D;
        probs[i] = A/abcd;
        probs[i+1] = B/abcd;
        probs[i+2] = C/abcd;
        probs[i+3] = D/abcd;
    }
    
    params.cudaDeviceNumEdges = nEdges ;
    params.cudaDeviceNumVertices = nVertices;
    params.cudaDeviceOutput = cudaDeviceOutput;
    params.cudaDeviceProbs = probs;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstGraphParams), &params, sizeof(GlobalConstants));

    return true;
}

bool destroy(){
    hipFree(cuConstGraphParams.cudaDeviceProbs);
    hipFree(cuConstGraphParams.cudaDeviceOutput);
    return true;
}
