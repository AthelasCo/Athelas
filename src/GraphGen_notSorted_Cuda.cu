#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#DEFINE MAX_DEPTH 128

__device__ inline int updiv(int n, int d) {
    return (n+d-1)/d;
}

static inline int updivHost(int n, int d) {
    return (n+d-1)/d;
}

struct GlobalConstants {

    unsigned long long cudaDeviceNumEdges, cudaDeviceNumVertices;
    double* cudaDeviceProbs;
    int* output;
};

__constant__ GlobalConstants cuConstGraphParams;

bool setup(
        const unsigned long long nEdges,
        const unsigned long long nVertices,
        const double RMAT_a, const double RMAT_b, const double RMAT_c,
        const unsigned int nCPUWorkerThreads,
        std::ofstream& outFile,
        const unsigned long long standardCapacity,
        const bool allowEdgeToSelf,
        const bool allowDuplicateEdges,
        const bool directedGraph,
        const bool sorted){
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 480") == 0
            || name.compare("GeForce GTX 670") == 0
            || name.compare("GeForce GTX 780") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 480, 670 or 780.\n");
        printf("---------------------------------------------------------\n");
    }
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy
    double* cudaDeviceOutput = NULL;
    hipMalloc(&cudaDeviceOutput, sizeof(double) * 2 * nEdges);

    GlobalConstants params;
    // // Initialize parameters in constant memory.  We didn't talk about
    // // constant memory in class, but the use of read-only constant
    // // memory here is an optimization over just sticking these values
    // // in device global memory.  NVIDIA GPUs have a few special tricks
    // // for optimizing access to constant memory.  Using global memory
    // // here would have worked just as well.  See the Programmer's
    // // Guide for more information about constant memory.


    //Generate Probabilities
    std::uniform_real_distribution<double> distribution(0.0,1.0);
    static std::default_random_engine generator;
    double probs[MAX_DEPTH*4]:
    for (int i = 0; i < MAX_DEPTH*4; i+=4) {
        double A = RMAT_a * (distribution(generator)+0.5);
        double B = RMAT_b * (distribution(generator)+0.5);
        double C = RMAT_c *(distribution(generator)+0.5);
        double D = (1- (RMAT_a+RMAT_b+RMAT_c)) *(distribution(generator)+0.5);
        double abcd = A+B+C+D;
        probs[i] = A/abcd;
        probs[i+1] = B/abcd;
        probs[i+2] = C/abcd;
        probs[i+3] = D/abcd;
    }
    
    params.cudaDeviceNumEdges = nEdges ;
    params.cudaDeviceNumVertices = nVertices;
    params.output = cudaDeviceOutput;
    params.cudaDeviceProbs = probs;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstGraphParams), &params, sizeof(GlobalConstants));

    return true;
}

bool destroy(){
    hipFree(cuConstGraphParams.cudaConstantProbTable);
    // hipFree(cuConstGraphParams);
}
