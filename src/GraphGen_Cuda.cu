#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <iostream>
#include <cstring>
#include <fstream>
#include <cstdlib>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "GraphGen_Cuda.h"
#include "internal_config.hpp"
#include "Square.hpp"
#include "Edge.hpp"
#include "utils.hpp"
#include "exclusiveScan.cu_inl"

#define cudacall(call) \
{ \
    hipError_t err = (call);                                                                                               \
    if(hipSuccess != err)                                                                                                  \
    {                                                                                                                       \
        fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
        hipDeviceReset();                                                                                                  \
        exit(EXIT_FAILURE);                                                                                                 \
    }                                                                                                                       \
} \

struct cudaSquare {
	uint X_start, X_end, Y_start, Y_end;
	uint nEdgeToGenerate, level, recIndex_horizontal, recIndex_vertical;
	uint thisEdgeToGenerate;
};

struct GlobalConstants {

    uint cudaDeviceNumEdges, cudaDeviceNumVertices;
    double* cudaDeviceProbs;
    int* cudaDeviceOutput;
    cudaSquare* cudaSquares;
    hiprandState_t* cudaThreadStates;
    int nSquares;
    bool directedGraph, allowEdgeToSelf, sorted;
};

__device__ inline int updiv(int n, int d) {
    return (n+d-1)/d;
}

__constant__ GlobalConstants cuConstGraphParams;

/* CUDA's random number library uses hiprandState_t to keep track of the seed value
   we will store a random state for every thread  */

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed) {

  /* we have to initialize the state */
    // printf("seed %d\n", seed);
  hiprandState_t* states = cuConstGraphParams.cudaThreadStates;
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x*blockDim.x+threadIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x*blockDim.x+threadIdx.x]);
  // const double RndProb = hiprand_uniform(states + blockIdx.x);
  // printf("RANDOM RANDOM %lf\n", RndProb);
}

__device__ __inline__ int2
get_Edge_indices(hiprandState_t* states,  uint offX, uint rngX, uint offY, uint rngY, double A[],double B[],double C[],double D[]) {
    uint x_offset = offX, y_offset = offY;
    uint rangeX = rngX, rangeY = rngY;
    uint depth =0;
    double sumA, sumAB, sumABC, sumAC;
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    hiprandState_t localState = states[idx];
    printf("reached here\n");
    while (rangeX > 1 || rangeY > 1) {
        printf("depth is %u\n",depth );
        // printf("%d %d\n",rngX,rngY );
        sumA = A[depth];
        sumAB = sumA + B[depth];
        sumAC = sumA + C[depth];
        sumABC = sumAB + C[depth];
        

        const double RndProb = hiprand_uniform(&localState);
        // printf("%d %d RANDOM %lf\n", blockIdx.x , threadIdx.x,RndProb );
        if (rangeX>1 && rangeY>1) {
          if (RndProb < sumA) { rangeX/=2; rangeY/=2; }
          else if (RndProb < sumAB) { x_offset+=rangeX/2;  rangeX-=rangeX/2;  rangeY/=2; }
          else if (RndProb < sumABC) { y_offset+=rangeY/2;  rangeX/=2;  rangeY-=rangeY/2; }
          else { x_offset+=rangeX/2;  y_offset+=rangeY/2;  rangeX-=rangeX/2;  rangeY-=rangeY/2; }
        } else
        if (rangeX>1) { // row vector
          if (RndProb < sumAC) { rangeX/=2; rangeY/=2; }
          else { x_offset+=rangeX/2;  rangeX-=rangeX/2;  rangeY/=2; }
        } else
        if (rangeY>1) { // column vector
          if (RndProb < sumAB) { rangeX/=2; rangeY/=2; }
          else { y_offset+=rangeY/2;  rangeX/=2;  rangeY-=rangeY/2; }
        } else{
            //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
        }
        depth++;
    }
    states[idx] = localState;
    int2 e;
    //printf("Edge %d %d\n", (int)x_offset, (int)y_offset);

    e.x = x_offset;
    e.y = y_offset;
    printf("returning here\n");

    return e;
}
__global__ void KernelGenerateEdges() {
    // std::uniform_int_distribution<>& dis, std::mt19937_64& gen,
    // std::vector<uint>& duplicate_indices
    //printf("BlockIdx %d ThreadIdx %d\n",blockIdx.x, threadIdx.x);
    hiprandState_t* states = cuConstGraphParams.cudaThreadStates;
    bool directedGraph = cuConstGraphParams.directedGraph;
    bool allowEdgeToSelf = cuConstGraphParams.allowEdgeToSelf;
    bool sorted = cuConstGraphParams.sorted;
    int blockIndex = blockIdx.x;
    int offset = blockIndex;
    int threadIndex = threadIdx.x;
    if (blockIndex < cuConstGraphParams.nSquares) {
        cudaSquare squ = (cudaSquare)cuConstGraphParams.cudaSquares[blockIndex];
        __shared__ uint offX;  
        __shared__ uint offY;  
        __shared__ uint rngX;  
        __shared__ uint rngY;  
        
        __shared__ uint nEdgesToGen;
        if (threadIndex==0)
        {
            offX = (uint)squ.X_start;
            offY = (uint)squ.Y_start;
            rngX = (uint)squ.X_end-offX;
            rngY = (uint)squ.Y_end-offY;
            nEdgesToGen = (uint)squ.nEdgeToGenerate;
            printf("Found Square x: [%u,%u] y: [%u, %u] %u\n", offX,  offX+rngX,offY,offY+rngY, nEdgesToGen);
        }   
        __shared__ double A[MAX_DEPTH];
        __shared__ double B[MAX_DEPTH];
        __shared__ double C[MAX_DEPTH];
        __shared__ double D[MAX_DEPTH];

        if (threadIndex==0)
        {
            for (int i = 0; i < MAX_DEPTH; ++i)
            {
                A[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i)]);
                B[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i) + 1]);
                C[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i)+ 2]);
                D[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i)+ 3]);
            }
            printf("ENDED probs\n");
        }
        __syncthreads();

        auto applyCondition = directedGraph || ( offX < offY); // true: if the graph is directed or in case it is undirected, the square belongs to the lower triangle of adjacency matrix. false: the diagonal passes the rectangle and the graph is undirected.


        unsigned maxIter = updiv(nEdgesToGen, blockDim.x);

        for (unsigned i = 0; i < maxIter; ++i)
        {
           int edgeIdx = i * blockDim.x + threadIndex;
           int2 e;
           if (edgeIdx < nEdgesToGen )
           {

               while(true) {
                   e = get_Edge_indices(states, offX, rngX, offY, rngY, A, B, C, D );
                   uint h_idx = e.x;
                   uint v_idx = e.y;
                   if( (!applyCondition && h_idx > v_idx) || (!allowEdgeToSelf && h_idx == v_idx ) ) {// Short-circuit if it doesn't pass the test.
                       printf("EdgeID %d fail1\n", edgeIdx );
                       continue;
                   } else if (h_idx< offX || h_idx>= offX+rngX || v_idx < offY || v_idx >= offY+rngY ){
                       printf("EdgeID %d recompute src %d dst %d tl %d tr %d bl %d br %d \n", edgeIdx, h_idx, v_idx, offX, offY, offX+rngX, offY+rngY);
                       break;
                   } else {
                       continue;
                   }
               }
               printf("Edges Calculated %d \t %d\n", e.x,e.y);
               cuConstGraphParams.cudaDeviceOutput[2*( squ.thisEdgeToGenerate + edgeIdx)] = e.x;
               cuConstGraphParams.cudaDeviceOutput[2*( squ.thisEdgeToGenerate + edgeIdx)+1] = e.y;

           }
           __syncthreads();
        }
        __syncthreads();
    }

}

////////////////////////////////////////////////////////////////////////////////////////
/* this GPU kernel function is used to initialize the random states */
__global__ void initSorted(unsigned int seed) {

  /* we have to initialize the state */
    // printf("seed %d\n", seed);
  hiprandState_t* states = cuConstGraphParams.cudaThreadStates;
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x*blockDim.x+threadIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x*blockDim.x+threadIdx.x]);
  // const double RndProb = hiprand_uniform(states + blockIdx.x);
  // printf("RANDOM RANDOM %lf\n", RndProb);
}

__device__ __inline__ int2
get_Edge_indices_PKSG(hiprandState_t* states, uint offX, uint rngX,uint offY, uint rngY, uint u, double A[],double B[],double C[],double D[]) {
    uint z=u, v=0, s=0;
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    printf("reached here\n");
    hiprandState_t localState = states[idx];
    // int k = ceil(log2((double)rngX));

    for (int depth = 0; rngX>0; ++depth, rngX/=2)
    {

        printf("depth %d\n",depth );
      double sumAB = A[depth] +B[depth];
      double a = A[depth]/sumAB;
      double b = B[depth]/sumAB;
      double c = C[depth]/(1-sumAB);
      double d = D[depth]/(1-sumAB);
      uint l = z%2;
      const double RndProb = hiprand_uniform(&localState);
      if (l==0) {
        s=1;
        if (RndProb<a) {
          s=0;
        }
      } else {
        s=1;
        if (RndProb<c) {
          s=0;   
        }
      }
      v= 2*v+s;
      z= z/2;
    }
    
    int2 e;
    e.x = u;
    e.y = v;
    return e;
}

__global__ void KernelGenerateEdgesPSKG() {
    hiprandState_t* states = cuConstGraphParams.cudaThreadStates;
    bool directedGraph = cuConstGraphParams.directedGraph;
    bool allowEdgeToSelf = cuConstGraphParams.allowEdgeToSelf;
    bool sorted = cuConstGraphParams.sorted;
    int blockIndex = blockIdx.x;
    int offset = blockIndex*6;
    int threadIndex = threadIdx.x;
    if (blockIndex < cuConstGraphParams.nSquares) {
        cudaSquare squ = (cudaSquare)cuConstGraphParams.cudaSquares[blockIndex];
        __shared__ uint offX;  
        __shared__ uint offY;  
        __shared__ uint rngX;  
        __shared__ uint rngY;  
        
        __shared__ uint nEdgesToGen;
        if (threadIndex==0)
        {
            offX = (uint)squ.X_start;
            offY = (uint)squ.Y_start;
            rngX = (uint)squ.X_end-offX;
            rngY = (uint)squ.Y_end-offY;
            nEdgesToGen = (uint)squ.nEdgeToGenerate;
            printf("Found Square %d with tl %d tr %d bl %d br %d and edges %d for tE %d\n", offset, offX, offY, offX+rngX, offY+rngY, nEdgesToGen, (uint)squ.thisEdgeToGenerate);        }   
        __shared__ double A[MAX_DEPTH];
        __shared__ double B[MAX_DEPTH];
        __shared__ double C[MAX_DEPTH];
        __shared__ double D[MAX_DEPTH];

        if (threadIndex==0)
        {
            for (int i = 0; i < MAX_DEPTH; ++i)
            {
                A[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i)]);
                B[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i) + 1]);
                C[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i)+ 2]);
                D[i] = (double)(cuConstGraphParams.cudaDeviceProbs[4 * (i)+ 3]);
            }
        }
        __syncthreads();
        int minN = min(NUM_CUDA_THREADS, (int)rngX);
        __shared__ uint shared_no_of_outdegs[NUM_CUDA_THREADS];
        __shared__ uint shared_output[NUM_CUDA_THREADS];
        volatile __shared__ uint shared_scratch[2 * NUM_CUDA_THREADS];

        auto applyCondition = directedGraph || ( offX < offY); // true: if the graph is directed or in case it is undirected, the square belongs to the lower triangle of adjacency matrix. false: the diagonal passes the rectangle and the graph is undirected.


        unsigned maxIter = updiv(rngX, blockDim.x); //Divide all sources amongst NUM_CUDA_THREADS
        int N = 2;

        for (unsigned i = 0; i < maxIter; ++i)
        {
            // shared_output[threadIdx.x] = 0;
            // shared_no_of_outdegs[threadIdx.x]= 0;
            int srcIdx = i * blockDim.x + threadIndex;//Interleave sources
            if (srcIdx < rngX+offX )
            {
                double p=nEdgesToGen;
                uint z = srcIdx;
                int j=0;
                uint localrngX = rngX;
                while(localrngX>0) {
                    uint l = z%N;
                    double Ul = A[j]+B[j];
                    if (l==1)
                    {
                      Ul = 1-(A[j]+B[j]);
                    }
                    p= p * Ul;
                    z = z/N;
                    localrngX/=2;
                    j++;
                }
                double ep =p;
                hiprandState_t localState = states[threadIndex];
                unsigned int X = hiprand_poisson(&localState, ep);
                shared_no_of_outdegs[threadIndex] = X;
                //Perform prefix_sum
                __syncthreads();
                sharedMemExclusiveScan(threadIndex, shared_no_of_outdegs, shared_output,
                              shared_scratch, minN);
                __syncthreads();
                //BUG: Manual sum of out degrees overflows net edges to generate
                //BUG: Prefix sum not working
                // printf("Found out degree %d for net out degree %d for nElements %d\n", X, shared_output[max(minN-1,0)], minN); 
                uint edgeIdx;
                for( edgeIdx = 0; edgeIdx < X ; ) {
                    int2 e;
                    e = get_Edge_indices_PKSG(states, offX, rngX, offY, rngY, srcIdx, A, B, C, D);
                    uint h_idx = e.x;
                    uint v_idx = e.y;
                    if( (!applyCondition && h_idx > v_idx) || (!allowEdgeToSelf && h_idx == v_idx ) ) {// Short-circuit if it doesn't pass the test.
                        printf("Err\n"); break;//continue;
                    //BUG: Code Hangs if below two lines included
                    //} else if (h_idx< offX || h_idx>= offX+rngX || v_idx < offY || v_idx >= offY+rngY ){
                    //    printf("Err2\n"); break;//continue;
                    } else {
                    printf("Edges Calculated %d \t %d\n", e.x,e.y);
                    ++edgeIdx;
                    //Write to file
                    }
                }
                printf("Generated %d edges in thread %d in block %d\n", edgeIdx, threadIndex, blockIdx.x );
             }
            __syncthreads();
        }
        __syncthreads();
    }

}

////////////////////////////////////////////////////////////////////////////////////////

GraphGen_Cuda::GraphGen_Cuda() {
    cudaDeviceProbs = NULL;
    cudaDeviceOutput = NULL;
    cudaDeviceSquares = NULL;
}


GraphGen_Cuda::~GraphGen_Cuda() {
    if (cudaDeviceProbs) {
        hipFree(cudaDeviceProbs);
        hipFree(cudaDeviceOutput);
        hipFree(cudaDeviceSquares);
        hipFree(cudaThreadStates);
   }
}

int GraphGen_Cuda::setup(
        const uint nEdges,
        const uint nVertices,
        const double RMAT_a, const double RMAT_b, const double RMAT_c,
        const uint standardCapacity,
        const bool allowEdgeToSelf,
        const bool allowDuplicateEdges,
        const bool directedGraph,
        const bool sorted
    ){
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy
    cudacall(hipMalloc(&cudaDeviceProbs, sizeof(double) * 4 * MAX_DEPTH));

    cudacall(hipMalloc(&cudaDeviceOutput, sizeof(int) * 2 * nEdges));

    GlobalConstants params;

    //Generate Probabilities
    std::uniform_real_distribution<double> distribution(0.0,1.0);
    static std::default_random_engine generator;
    double probs[MAX_DEPTH*4];
    for (int i = 0; i < MAX_DEPTH*4; i+=4) {
        double A = RMAT_a * (distribution(generator)+0.5);
        double B = RMAT_b * (distribution(generator)+0.5);
        double C = RMAT_c *(distribution(generator)+0.5);
        double D = (1- (RMAT_a+RMAT_b+RMAT_c)) *(distribution(generator)+0.5);
        double abcd = A+B+C+D;
        probs[i] = A/abcd;
        probs[i+1] = B/abcd;
        probs[i+2] = C/abcd;
        probs[i+3] = D/abcd;
    }
    
    params.cudaDeviceNumEdges = nEdges ;
    params.cudaDeviceNumVertices = nVertices;
    params.cudaDeviceOutput = cudaDeviceOutput;
    hipMemcpy(cudaDeviceProbs, probs, sizeof(double) * 4 * MAX_DEPTH, hipMemcpyHostToDevice);
    params.cudaDeviceProbs = cudaDeviceProbs;

    //Generate Squares
    std::vector<Square> squares ( 1, Square( 0, nVertices, 0, nVertices, nEdges, 0, 0, 0 ) );
	bool allRecsAreInRange;
	do {
		allRecsAreInRange = true;

		unsigned int recIdx = 0;
		for( auto& rec: squares ) {

			if( Eligible_RNG_Rec(rec, standardCapacity) ) {
				// continue;
			} else {
				ShatterSquare(squares, RMAT_a, RMAT_b, RMAT_c, recIdx, directedGraph);
				allRecsAreInRange = false;
				
				break;
			}
			++recIdx;
		}
	} while( !allRecsAreInRange );

	// Making sure there are enough squares to utilize all blocks and not more
	while( squares.size() < NUM_BLOCKS && !edgeOverflow(squares) ) {
		// Shattering the biggest rectangle.
		uint biggest_size = 0;
		unsigned int biggest_index = 0;
		for( unsigned int x = 0; x < squares.size(); ++x )
			if( squares.at(x).getnEdges() > biggest_size ) {
				biggest_size = squares.at(x).getnEdges();
				biggest_index = x;
			}
		ShatterSquare(squares, RMAT_a, RMAT_b, RMAT_c, biggest_index, directedGraph);
	}

	if (allowDuplicateEdges)
	{
		int originalSize = squares.size();
		for (int index = 0; index < originalSize; ++index)
		{
			//memory leak?
			Square srcRect(squares.at(index));
			// squares.erase(squares.begin()+index);
		
			int numEdgesAssigned = 0;
			int edgesPerSquare = srcRect.getnEdges()/NUM_BLOCKS;
			if (edgesPerSquare<20000)
			{
				continue;
			}
			for( unsigned int i = 0; i < NUM_BLOCKS-1; ++i ){
				Square destRect(srcRect);
				destRect.setnEdges(edgesPerSquare);
				numEdgesAssigned+=edgesPerSquare;
				squares.push_back(destRect);

			}
			srcRect.setnEdges( srcRect.getnEdges()-numEdgesAssigned);
			squares.at(index) = srcRect;
		}

	
	}
	std::sort(squares.begin(), squares.end(),std::greater<Square>());

    //uint* allSquares = (uint*) malloc(sizeof(uint)* 6 * squares.size());
    cudaSquare* allSquares = (cudaSquare*) malloc(sizeof(cudaSquare) * squares.size());
    printf("Generated Squres\n");

    uint tEdges = 0;

    for( unsigned int x = 0; x < squares.size(); ++x ) {
		Square& rec = squares.at( x );
        cudaSquare newSquare;
        newSquare.X_start = rec.get_X_start();
        newSquare.X_end = rec.get_X_end();
        newSquare.Y_start = rec.get_Y_start();
        newSquare.Y_end = rec.get_Y_end();
        newSquare.nEdgeToGenerate = rec.getnEdges();
        newSquare.level = 0;//TODO
        newSquare.recIndex_horizontal = rec.get_H_idx();
        newSquare.recIndex_vertical = rec.get_V_idx();
        newSquare.thisEdgeToGenerate = tEdges;
        memcpy(allSquares+x, &newSquare, sizeof(cudaSquare));
        //*(allSquares + 6*x) = rec.get_X_start();
        //printf("Found sq start %d %d\n", rec.get_X_start(), *(allSquares+6*x));
        //*(allSquares + 6*x + 1)  = rec.get_X_end();
        //*(allSquares + 6*x + 2)  = rec.get_Y_start();
        //*(allSquares + 6*x + 3)  = rec.get_Y_end();
        //printf("Found edges %d\n", rec.getnEdges());
        //*(allSquares + 6*x + 4)  = rec.getnEdges();
        //*(allSquares + 6*x + 5)  = tEdges;
        tEdges += rec.getnEdges();
    }
    printf("Copying Squres\n");
    cudacall(hipMalloc(&cudaDeviceSquares, sizeof(cudaSquare) * squares.size()));
    cudacall(hipMemcpy(cudaDeviceSquares, allSquares, sizeof(cudaSquare) * squares.size(), hipMemcpyHostToDevice));
    params.cudaSquares = cudaDeviceSquares;
    params.nSquares = squares.size();

    /* allocate space on the GPU for the random states */
    cudacall(hipMalloc((void**) &cudaThreadStates, squares.size()*NUM_CUDA_THREADS * sizeof(hiprandState_t)));
    params.cudaThreadStates = cudaThreadStates;
    params.allowEdgeToSelf = allowEdgeToSelf;
    params.directedGraph = directedGraph;
    params.sorted = sorted;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstGraphParams), &params, sizeof(GlobalConstants));
    /* invoke the GPU to initialize all of the random states */
    initSorted<<<squares.size(), NUM_CUDA_THREADS>>>(time(0));
    hipDeviceSynchronize();

    for( unsigned int x = 0; x < squares.size(); ++x ){
        std::cout << squares.at(x);
    }
    std::cout << "CUDA Error " << hipGetErrorString(hipGetLastError()) << "\n";
    free(allSquares);
    return squares.size();
}

void GraphGen_Cuda::generate(const bool directedGraph,
        const bool allowEdgeToSelf, const bool sorted, int squares_size) {
    dim3 nThreads(NUM_CUDA_THREADS,1,1);
    // dim3 gridDim(updivHost(squares_size, blockDim.x));
    dim3 nBlocks(squares_size,1,1);
    printf("Hello launching kernel of blocks %d %d %d and tpb %d %d %d\n", nBlocks.x, nBlocks.y, nBlocks.z, nThreads.x, nThreads.y, nThreads.z);
    KernelGenerateEdges<<<nBlocks, nThreads>>>();
    hipDeviceSynchronize();
    std::cout << "CUDA Error " << hipGetErrorString(hipGetLastError());
    
    printf("Bye \n");

}

uint GraphGen_Cuda::printGraph(unsigned *Graph, uint nEdges, std::ofstream& outFile) {
    uint x;
    for (x = 0; x < nEdges; x++) {
         outFile << Graph[2*x] << "\t" << Graph[2*x+1] << "\n";
    }
    return x;
}

bool GraphGen_Cuda::destroy(){
    //hipFree(states);
    hipFree(cudaDeviceProbs);
    hipFree(cudaDeviceOutput);
    return true;
}

void GraphGen_Cuda::getGraph(unsigned* Graph, uint nEdges) {
     hipMemcpy(Graph, cudaDeviceOutput, sizeof(int)*2*nEdges, hipMemcpyDeviceToHost);
}

